#include <iostream>
#include <fstream>
#include <string>
#include <cstdio>
#include <stdlib.h>
#include "hip/hip_runtime.h"
#include ""

using namespace std;


__device__ __host__ std::pair<uint64_t,uint64_t> strtouint64(auto* password_text){
	uint64_t uword1 = 0;
	uint64_t uword2 = 0;
	// uint64_t uword3 = 0;
	auto password_str = std::string(password_text);
	
	for(int i = 0; i < password_str.length(); i++)
	{
		uint8_t uchar = (uint8_t) (int) password_str[i];
		if (i < 8) 
		{
			uword1 = (uword1 << 8) | uchar;
		}
		else
		{
			uword2 = (uword2 << 8) | uchar;
		}
	}
	return {uword1,uword2};
}
